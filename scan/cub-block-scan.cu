#include "hip/hip_runtime.h"
#include "scan.cuh"
using namespace hipcub;

template <
    int                 TPB,
    int                 IPT,
    BlockScanAlgorithm  ALGORITHM>
__global__ void BlockPrefixSumKernel_cub(float *out, float *in) {
    typedef BlockLoad<float, TPB, IPT, BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;
    typedef BlockStore<float, TPB, IPT, BLOCK_STORE_WARP_TRANSPOSE> BlockStoreT;
    typedef BlockScan<float, TPB, ALGORITHM> BlockScanT;

    __shared__ union TempStorage {
        typename BlockLoadT::TempStorage    load;
        typename BlockStoreT::TempStorage   store;
        typename BlockScanT::TempStorage    scan;
    } temp_storage;

    float data[IPT];
    BlockLoadT(temp_storage.load).Load(in, data);
    __syncthreads();
    
    float aggregate;
    BlockScanT(temp_storage.scan).ExclusiveSum(data, data, aggregate);
    __syncthreads();

    BlockStoreT(temp_storage.store).Store(out, data);

    if (threadIdx.x == 0) out[TPB * IPT] = aggregate;
}

void init_h(float *&h, int num) {
    for(int ii = 0; ii < num; ii++) 
        h[ii] = float(ii);
}

int main(int argc, char **argv) {
    int numElements = 4096;
    if(argc > 1) numElements = std::atoi(argv[1]);
    float *d_in = nullptr;
    float *d_out = nullptr;
    float *h_in = new float [numElements];
    float *h_out = new float [numElements + 1];
        
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_in, numElements * sizeof(float));
    hipMalloc(&d_out, (numElements + 1) * sizeof(float));
    init_h(h_in, numElements);
    cuErrCheck(hipMemcpy(d_in, h_in, sizeof(float) * numElements, hipMemcpyHostToDevice));
    
    hipEventRecord(start);
    if (numElements == 256) {
        BlockPrefixSumKernel_cub<TPB64, 4, BLOCK_SCAN_RAKING><<<1, TPB64>>>(
            d_out,
            d_in);
    }
    else if (numElements == 1024) {
        BlockPrefixSumKernel_cub<TPB256, 4, BLOCK_SCAN_RAKING><<<1, TPB256>>>(
            d_out,
            d_in);
    }
    else if (numElements == 4096) {
        BlockPrefixSumKernel_cub<TPB1024, 4, BLOCK_SCAN_RAKING><<<1, TPB1024>>>(
            d_out,
            d_in);
    }
    cuErrCheck(hipGetLastError());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Cub Scan %i elements takes %f ms\n", numElements, milliseconds);
    cuErrCheck(hipMemcpy(h_out, d_out, sizeof(float) * (numElements + 1), hipMemcpyDeviceToHost));
    printf("Result  :  sum = %f\n", h_out[numElements-1]);

    hipFree(d_in);
    hipFree(d_out);
    delete[] h_in;
    delete[] h_out;
    return 0;
}
